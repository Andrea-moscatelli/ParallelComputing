
#include <hip/hip_runtime.h>
#include <algorithm>
#include <ctime>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <iostream>
#include <list>
#include <map>
#include <numeric>
#include <sstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


bool FIND_BIGRAM = false; //true = find brigrams; false = find trigrams
bool PRINT = false; //if set to true it will print the found bigrams and trigrams
int GRID_DIM = 10; // grid size
int BLOCK_DIM = 128; //block size
std::string nameFile = "inputTextLong.txt"; //the name of the text file to analyse


// this utility method allows the user to better understand the CUDA errors
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

// converts the passed text line into only lower case alphabet characters
__host__ string clean(string in) {
	string final;
	for(int i = 0; i < in.length(); i++) {
		if(isalpha(in[i])) final += tolower(in[i]);
	}
	return final;
}

// this method finds the graphems (bigram or trigram) using the CPU
__host__ void findGraphemsWithCPU(string line, std::map<std::string,int> &graphems) {

	int tail = FIND_BIGRAM? 1 : 2;

	for(int i = 0; i < line.length()-tail; i++) {

		string key = std::string() + line[i] + line[i+1];
		if(!FIND_BIGRAM)
			key = key + line[i+2];

		std::map<std::string,int>::iterator it = graphems.find(key);
		if(it != graphems.end()){
			it->second++;
		}else{
			graphems.insert(std::make_pair(key, 1));
		}

	}

}

// this method finds the graphems (bigram or trigram) using the CPU
__host__ std::map<std::string,int> methodWithCPU(std::string line){
	std::map<std::string,int> graphems;

	findGraphemsWithCPU(line,graphems);

	return graphems;

}

// this method converts a character into an int
__device__ int getCharIndex(char c){
	return (c - 'a');
}

//this method finds the graphems (bigram or trigram) using the GPU
__global__ void findGraphemsWithGPU(const char *line, int* graphemsArray, int sliceLength, int lineLength, bool findBigram) {

	int startPoint =
			blockDim.x * blockIdx.x +
			threadIdx.x;

	startPoint *= sliceLength;

	int endPoint = startPoint + sliceLength - 1;
	int tail = findBigram? 1 : 2;
	endPoint += tail;

	int index1;
	int index2;
	int index3;
	if((startPoint+tail) < lineLength ){
		index2 = getCharIndex(line[startPoint]);
		if(!findBigram) {
			index3 = getCharIndex(line[startPoint+1]);
		}
	}


	while((startPoint+tail) <= endPoint && (startPoint+tail) < lineLength){
		index1 = index2;
		if(findBigram) {
			index2 = getCharIndex(line[startPoint+tail]);
			atomicAdd(&graphemsArray[index1 * 26 + index2 ], 1);
		}else{
			index2 = index3;
			index3 = getCharIndex(line[startPoint+tail]);
			atomicAdd(&graphemsArray[index1 * 26 * 26 + index2 * 26 + index3], 1);
		}

		startPoint++;
	}

	return;
}

// this method prints the graphems found with the GPU
__host__ void print(int *graphemsArrayHost){
	int lengthGraphems = FIND_BIGRAM? 26*26 : 26*26*26;
	std::string alphabet = "abcdefghijklmnopqrstuvwxyz";
	for(int i = 0 ; i < lengthGraphems; i++){
		if(graphemsArrayHost[i] != 0){
			div_t result1 = std::div(i,26);
			div_t result2 = std::div(result1.quot,26);
			if(FIND_BIGRAM){
				cout << (std::string() + alphabet[result2.rem]+ alphabet[result1.rem]) << " = " << graphemsArrayHost[i] << "\n";
			}else{
				div_t result3 = std::div(result2.quot,26);
				cout << (std::string() + alphabet[result3.rem]+ alphabet[result2.rem] + alphabet[result1.rem]) << " = " << graphemsArrayHost[i] << "\n";
			}
		}
	}
}

// this method finds the graphems (bigram or trigram) using the GPU
__host__ int* methodWithGPU(std::string line){

	// GRAPHEMS ARRAY
	int lengthGraphems = FIND_BIGRAM? 26*26 : 26*26*26;
	int *graphemsArrayDevice;
	int *graphemsArrayHost=(int*)calloc(lengthGraphems,sizeof(int));


	//	allocate device memory
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&graphemsArrayDevice,
					sizeof(int) * lengthGraphems));

	//	copy from host to device memory
	CUDA_CHECK_RETURN(
			hipMemcpy(graphemsArrayDevice, graphemsArrayHost, lengthGraphems * sizeof(int),
					hipMemcpyHostToDevice));

	//  TEXT LINE
	int lengthLine = line.length();
	char *lineDevice;

	//  allocate device memory
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&lineDevice,
					sizeof(char) * lengthLine));
	//
	//	copy from host to device memory
	CUDA_CHECK_RETURN(
			hipMemcpy(lineDevice, line.c_str(), lengthLine * sizeof(char),
					hipMemcpyHostToDevice));


	// execute kernel
	int totalthreadNumber = GRID_DIM * BLOCK_DIM;
	int sliceLength = ceil(float(lengthLine)/float(totalthreadNumber));
	findGraphemsWithGPU<<< GRID_DIM, BLOCK_DIM >>>(lineDevice, graphemsArrayDevice, sliceLength, lengthLine, FIND_BIGRAM);
	//
	hipDeviceSynchronize();

	//	copy results from device memory to host
	CUDA_CHECK_RETURN(
			hipMemcpy(graphemsArrayHost, graphemsArrayDevice, lengthGraphems * sizeof(int),
					hipMemcpyDeviceToHost));


	// Free the GPU memory here
	hipFree(lineDevice);
	hipFree(graphemsArrayDevice);
	return graphemsArrayHost;

}

// The main method.
// Parameters:
// 1 - [b,t] in order to chose between "Bigrams" or "Trigrams" (default: b)
// 2 - size of grid for the initial call (default: 10)
// 3 - size of block for the initial call (default: 128)
// 4 - [t,f,true,false] to print the result of the graphems (default: false)
// 5 - the name of the input file (default: inputTextLong.txt)
//
// calling example: ./main t 5 32 true inputTextVeryLong.txtx
__host__ int main(int argc, char** argv) {
	if(argc > 1){
		std::string setting(argv[1]);
		if(setting == "b" ) {
			FIND_BIGRAM = true;
		}else if(setting == "t" ) {
			FIND_BIGRAM = false;
		}else{
			cout<<"Parameter "<< argv[1] <<" not accepted. Only \"b\" (bigram), \"t\" (trigram), accepted. "<< "\n";
			return 0;
		}
		if(argc > 2){
			GRID_DIM = atoi(argv[2]);
			if(argc > 3){
				BLOCK_DIM = atoi(argv[3]);
				if(argc > 4){
					std::string setting(argv[4]);
					if (setting == "t" || setting == "true")
						PRINT = true;
					if(argc > 5){
						std::string setting(argv[5]);
						nameFile = setting;
					}
				}
			}
		}
	}


	std::string line;
	std::string longLine;
	std::string path = "input/"+nameFile;
	ifstream myfile(path.c_str());
	if (myfile.is_open()) {
		while (getline(myfile, line)) {
			//	Cleaning the line
			line = clean(line);
			longLine += line;

		}
		myfile.close();
	}

	else
		cout << "Unable to open file";


	clock_t beginCPU = clock();
	std::map<std::string,int> graphems;
	graphems = methodWithCPU(longLine);
	clock_t endCPU = clock();

	//	showing contents:
	cout<< "GRID_DIM: " << GRID_DIM << ", BLOCK_DIM: " << BLOCK_DIM << "\n";
	double elapsed_secsCPU = double(endCPU - beginCPU) / CLOCKS_PER_SEC;
	cout<<"CPU METHOD: " << "\n";
	cout<<"Elapsed milliseconds: " << elapsed_secsCPU*1000 << "\n";
	cout<<"Microseconds: " << endCPU - beginCPU << "\n";

	// ITERATION TO START COMUNICATION WITH GPU
	int *graphemsArrayHost;
	clock_t beginGPU = clock();
	graphemsArrayHost = methodWithGPU(longLine);
	clock_t endGPU = clock();
	// Free host memory
	double elapsed_secsGPU =  double(endGPU - beginGPU) / CLOCKS_PER_SEC;
	std::cout << "FIRST ITERATION. GRID_DIM: " << GRID_DIM << ", BLOCK_DIM: " << BLOCK_DIM << "\n";
	std::cout << "Elapsed Milliseconds: " << elapsed_secsGPU*1000 << "\n";

	//verify data
	if(PRINT){
		std::cout << "The graphems obtained with CPU are:\n";
		std::map<std::string,int>::iterator it;
		for (it=graphems.begin(); it!=graphems.end(); ++it)
			std::cout << it->first << " => " << it->second << '\n';

		std::cout << "\n\n -----------------------------------------\n\n";
		std::cout << "The graphems obtained with GPU are:\n";
		print(graphemsArrayHost);
	}
	free(graphemsArrayHost);


	std::cout << "Elapsed milliseconds changing grid dimension and block dimension: \n";

	for (int dimBlocco=1; dimBlocco <= 512 ; dimBlocco = dimBlocco*2 ){
		std::cout << "," << dimBlocco;
	}


	std::cout << "\n\n";
	for (int dimGriglia=1; dimGriglia <= 512 ; dimGriglia = dimGriglia*2 ){
		GRID_DIM = dimGriglia;
		std::cout << dimGriglia;
		for (int dimBlocco=1; dimBlocco <= 512 ; dimBlocco = dimBlocco*2 ){
			BLOCK_DIM = dimBlocco;
			int *graphemsArrayHost;
			clock_t beginGPU = clock();
			graphemsArrayHost = methodWithGPU(longLine);
			clock_t endGPU = clock();

			// Free host memory
			free(graphemsArrayHost);

			double elapsed_secsGPU =  double(endGPU - beginGPU) / CLOCKS_PER_SEC;
			std::cout << ", "<< elapsed_secsGPU*1000 ;
		}
		std::cout <<  "\n";
	}
	return 0;
}
